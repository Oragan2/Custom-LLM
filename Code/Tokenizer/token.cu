#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <unordered_map>
#include <map>
#include <string>
#include <hip/hip_runtime.h>
#include <thrust/extrema.h>
#include <thrust/copy.h>

// Global variables and constants

std::vector<uint64_t> merges; // Store the merges
std::map<std::string, int> vocab; // Store the tokens and their ids
std::map<int, std::string> i_vocab; // Store the ids and their token
int batch_size = 1024 * 1024 * 4; // 4MB batch
std::vector<std::vector<uint32_t>> corpus_batches; // Store the batches of the corpus
// GPU variable
uint32_t* d_corpus;
uint32_t* d_hash_t;
uint64_t* d_pairs_table;
uint32_t* new_corpus;
uint32_t* place;
const size_t table_size = 200000000; // number of slots
int new_token_id = 0;
struct is_one {
    __host__ __device__
    bool operator()(const int x) const {
        return x == 1;
    }
};


// GPU functions declarations

__device__ inline uint64_t hash64(uint64_t key) {
	key ^= key >> 33;
	key *= 0xff51afd7ed558ccdULL;
	key ^= key >> 33;
	key *= 0xc4ceb9fe1a85ec53ULL;
	key ^= key >> 33;
	return key;
}

__global__ void GeneratePairs(uint32_t* corpus, uint32_t* hashs, uint64_t* hash_p, size_t N) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	const size_t table_size = 200000000; // number of slots

	for (size_t i = idx; i < N - 1; i += stride) {
		uint64_t pair = ((uint64_t)corpus[i] << 32) | (uint64_t)corpus[i + 1];
		uint64_t hash_idx = hash64(pair) % table_size;
		uint64_t old = atomicAdd(&hashs[hash_idx], 1);

		if (old == 0) {
			hash_p[hash_idx] = pair;
		}
	}
}

__global__ void findBest(uint32_t* corpus, uint64_t best_pair, uint32_t* place, int N) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (size_t i = idx; i < N - 1; i += stride) {
		uint64_t pair = ((uint64_t)corpus[i] << 32) | (uint64_t)corpus[i + 1];
		if (pair == best_pair) {
			place[i] = 1;
		}
		else {
			place[i] = 0;
		}
	}
}

__global__ void updateHash() {
	
}

// CPU helper functions declarations

void write() {
	std::ofstream merge_file("merges.txt", std::ios::app);
	for (auto& p : merges) {
		merge_file << (int)(uint32_t)(p >> 32) << " " << (int)(uint32_t)(p & 0xFFFFFFFF) << "\n";
	}
	merge_file.close();
	std::ofstream vocab_file("vocab.json", std::ios::app);
	for (auto it = vocab.begin(); it != vocab.end(); ) {
		vocab_file << "  \"" << it->first << "\": " << it->second;
		if (++it != vocab.end()) vocab_file << ",";
		vocab_file << "\n";
	}
	vocab_file.close();
}

void hash_table() {
	/// Create the pairs and their counts
	hipEvent_t start, stop;
	float milliseconds = 0;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Start timing
	hipEventRecord(start);

	for (auto& c : corpus_batches) {
		// Copy batch to device
		size_t N = c.size();
		hipMemcpy(d_corpus, c.data(), N * sizeof(uint32_t), hipMemcpyHostToDevice);

		// Define kernel launch parameters
		int blockSize = 256;
		int numBlocks = (N + blockSize - 1) / blockSize;

		// Launch kernel to generate pairs
		GeneratePairs << <numBlocks, blockSize >> > (d_corpus, d_hash_t, d_pairs_table, N);
		hipError_t err = hipGetLastError();
		if (err != hipSuccess)
			std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;

		// Wait for GPU to finish
		hipDeviceSynchronize();
	}

	// Stop timing
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "Time to build hash table: " << milliseconds << " ms" << std::endl;
	
	// Cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

uint64_t maxi() {
	hipEvent_t start, stop;
	float milliseconds = 0;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);

	auto max_it = thrust::max_element(thrust::device, d_hash_t, d_hash_t + table_size);
	int max_idx = max_it - d_hash_t;

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "Time to find max_element: " << milliseconds << " ms" << std::endl;

	// Cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy hash table back to host
	uint64_t host_most_frequent_pair;
	hipMemcpy(&host_most_frequent_pair, &d_pairs_table[max_idx], sizeof(uint64_t), hipMemcpyDeviceToHost);
	vocab.emplace(i_vocab[(int)(uint32_t)host_most_frequent_pair >> 32]+i_vocab[(int)(uint32_t)host_most_frequent_pair & 0xFFFFFFFF], new_token_id++);
	return host_most_frequent_pair;
}

void merge(uint64_t most_frequent_pair) {
	hipEvent_t start, stop;
	float milliseconds = 0;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Start timing
	hipEventRecord(start);
	for (auto& c : corpus_batches) {
			// Copy batch to device
			size_t N = c.size();
			hipMemcpy(d_corpus, c.data(), N * sizeof(uint32_t), hipMemcpyHostToDevice);

			// Define kernel launch parameters
			int blockSize = 256;
			int numBlocks = (N + blockSize - 1) / blockSize;

			// Launch kernel to generate pairs
			findBest << <numBlocks, blockSize >> > (d_corpus, most_frequent_pair, place , N);
			hipError_t err = hipGetLastError();
			if (err != hipSuccess)
				std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;

			// Wait for GPU to finish
			hipDeviceSynchronize();
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "Time to build hash table: " << milliseconds << " ms" << std::endl;
	
	// Cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

// main
int main() {
	// Load the file
	std::ifstream text8("text8", std::ios::binary);
	if (!text8) {
		std::cerr << "Error opening text8 file" << std::endl;
		return 1;
	}
	std::vector<char> raw((std::istreambuf_iterator<char>(text8)), {});
	text8.close();
	size_t corpus_size = raw.size();
	std::cout << "Read " << corpus_size << " bytes\n";
	// Split the file into batches
	std::vector<std::vector<char>> batches;
	std::cout << "Creating batches of size " << batch_size << " bytes\n";
	for (size_t i = 0; i < corpus_size; i += batch_size) {
		size_t end = std::min(i + batch_size, corpus_size);
		batches.emplace_back(raw.begin() + i, raw.begin() + end); // Slice the raw data into batches
	}
	// Turn the ASSCI into uint8
	for (auto& batch : batches) {
		std::vector<uint32_t> corpus(batch.size());
		for (size_t i = 0; i < batch.size(); i++) {
			corpus[i] = (uint32_t)batch[i];
		}
		corpus_batches.push_back(std::move(corpus));
	}

	std::cout << "Created " << corpus_batches.size() << " batches\n";

	int vocab_size = 500; // Desired vocab size

	// Initialize the vocab with single characters

	vocab.emplace("<EOS>", new_token_id++); // End of sequence token
	vocab.emplace("<UNK>", new_token_id++); // Unknown token
	vocab.emplace("<CODE>", new_token_id++); // Code token
	vocab.emplace("<FR>", new_token_id++); // French token
	vocab.emplace("<EN>", new_token_id++); // English token
	for (int i = 32; i < 256; i++) {
		std::string s(1, (char)i);
		vocab.emplace(s, i);
		i_vocab.emplace(i, s);
	}
	new_token_id = 257; // Reset new_token_id to 257 for me to have token 0-32 for special ones

	hipMalloc(&d_corpus, batch_size * sizeof(uint32_t));
	hipMalloc(&d_hash_t, table_size * sizeof(uint32_t));
	hipMalloc(&d_pairs_table, table_size * sizeof(uint64_t));
	hipMalloc(&new_corpus, batch_size * sizeof(uint32_t));
	hipMalloc(&place, batch_size * sizeof(uint32_t));

	hash_table();	

	while(new_token_id < vocab_size) {
		uint64_t most_frequent_pair = maxi();
		merge(most_frequent_pair);
		write();
	}

	hipFree(&d_corpus);
	hipFree(&d_hash_t);
	hipFree(&d_pairs_table);
	hipFree(&new_corpus);
	hipFree(&place);
	return 0;
}