#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include <vector>
#include <random>
#include <algorithm>
#include <stdexcept>
#include <limits>
#include <map>
#include <hip/hip_runtime.h>
#include "transformer.hpp"

// GPU functions

// CPU functions
// Helper
std::vector<std::vector<float>> initialize_matrix(int rows, int cols)
{
    float limit = 0.1f;
    std::vector<std::vector<float>> mat(rows, std::vector<float>(cols));
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(-limit, limit);

    for (int i = 0; i < rows; i++)
        for (int j = 0; j < cols; j++)
            mat[i][j] = dist(gen);

    return mat;
}

std::vector<std::vector<float>> matmul(const std::vector<std::vector<float>>& A, const std::vector<std::vector<float>>& B)
{
    int a_rows = A.size();
    int a_cols = A[0].size();
    int b_rows = B.size();
    int b_cols = B[0].size();

    if (a_cols != b_rows)
        throw std::invalid_argument("Incompatible matrix dimensions for multiplication.");

    std::vector<std::vector<float>> C(a_rows, std::vector<float>(b_cols, 0.0f));

    for (int i = 0; i < a_rows; i++)
        for (int j = 0; j < b_cols; j++)
            for (int k = 0; k < a_cols; k++)
                C[i][j] += A[i][k] * B[k][j];

    return C;
}

std::vector<std::vector<float>> softmax(const std::vector<std::vector<float>>& mat)
{
    std::vector<std::vector<float>> result = mat;
    for (auto& row : result)
    {
        float max_val = *std::max_element(row.begin(), row.end());
        float sum = 0.0f;
        for (auto& val : row)
        {
            val = std::exp(val - max_val);
            sum += val;
        }
        for (auto& val : row)
            val /= sum;
    }
    return result;
}

std::vector<std::vector<float>> transpose(const std::vector<std::vector<float>>& mat) {
    int rows = mat.size();
    int cols = mat[0].size();
    std::vector<std::vector<float>> transposed(cols, std::vector<float>(rows));
    for (int i = 0; i < rows; i++)
        for (int j = 0; j < cols; j++)
            transposed[j][i] = mat[i][j];
    return transposed;
}

MultiHeadAttention::MultiHeadAttention(int h, int d) : num_heads(h), hidden_dim(d)
{
    head_dim = hidden_dim / num_heads;
    W_Q = initialize_matrix(hidden_dim, hidden_dim);
    W_K = initialize_matrix(hidden_dim, hidden_dim);
    W_V = initialize_matrix(hidden_dim, hidden_dim);
    W_O = initialize_matrix(hidden_dim, hidden_dim);
}

std::vector<std::vector<float>> MultiHeadAttention::forward(const std::vector<std::vector<float>>& X) {
    // 1) projections
    std::vector<std::vector<float>> Q = matmul(X, W_Q);
    std::vector<std::vector<float>> K = matmul(X, W_K);
    std::vector<std::vector<float>> V = matmul(X, W_V);

    int seq_len = X.size();
    int head_dim = hidden_dim / num_heads;

    // 2) split heads
    std::vector<std::vector<float>> concat(seq_len, std::vector<float>(hidden_dim, 0.0f));
    for (int h = 0; h < num_heads; h++) {
        // slice per head
        std::vector<std::vector<float>> Qh(seq_len, std::vector<float>(head_dim));
        std::vector<std::vector<float>> Kh(seq_len, std::vector<float>(head_dim));
        std::vector<std::vector<float>> Vh(seq_len, std::vector<float>(head_dim));
        for (int i = 0; i < seq_len; i++) {
            for (int j = 0; j < head_dim; j++) {
                Qh[i][j] = Q[i][h * head_dim + j];
                Kh[i][j] = K[i][h * head_dim + j];
                Vh[i][j] = V[i][h * head_dim + j];
            }
        }
        // 3) attention: softmax((QK^T)/sqrt(dk)) * V
        auto scores = matmul(Qh, transpose(Kh));
        for (auto& row : scores) for (auto& v : row) v /= std::sqrt((float)head_dim);
        auto attn = softmax(scores);
        auto head_out = matmul(attn, Vh);

        // 4) copy into concat
        for (int i = 0; i < seq_len; i++) {
            for (int j = 0; j < head_dim; j++) {
                concat[i][h * head_dim + j] = head_out[i][j];
            }
        }
    }

    // 5) final projection
    return matmul(concat, W_O);
}

LLM::LLM(int vocab_size, int max_seq_len, int hidden_dim, int num_head)
    : embeding(vocab_size, std::vector<float>(hidden_dim, 0.0f)),
      positionalEncoding(sinusoidalEncoding(max_seq_len, hidden_dim)),
      transformer(num_head, hidden_dim){}

// temp on the CPU for testing the logic
std::vector<std::vector<float>> LLM::tokenEmbeding(std::vector<int>& tokens)
{
    size_t seq_len = tokens.size();
    size_t hidden_dim = embeding[0].size();

    std::vector<std::vector<float>> x(seq_len, std::vector<float>(hidden_dim, 0.0f));

    for (size_t i = 0; i < seq_len; i++)
    {
        int token_id = tokens[i];
        for (size_t d = 0; d < hidden_dim; d++)
        {
            x[i][d] = embeding[token_id][d] + positionalEncoding[i][d];
        }
    }
    return x;
}

// temp on the CPU for testing logic
std::vector<std::vector<float>> LLM::forwardPass(std::string text) {
    std::vector<int> tokens = {286,5012,3795};
    // TODO : make a real tokenizer
    auto X = tokenEmbeding(tokens);
    return transformer.forward(X);
}

// temp will be changed later
std::vector<std::vector<float>> LLM::sinusoidalEncoding(int seq_len, int dim)
{
    std::vector<std::vector<float>> pe(seq_len, std::vector<float>(dim));

    for (int pos = 0; pos < seq_len; pos++)
    {
        for (int i = 0; i < dim; i++)
        {
            double angle = pos / std::pow(10000.0, (2 * (i / 2)) / (double)dim);
            if (i % 2 == 0)
                pe[pos][i] = std::sin(angle);
            else
                pe[pos][i] = std::cos(angle);
        }
    }
    return pe;
}